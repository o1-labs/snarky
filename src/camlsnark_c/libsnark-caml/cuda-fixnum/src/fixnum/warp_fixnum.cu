#include "hip/hip_runtime.h"
#pragma once

#include "slot_layout.cu"
#include "word_fixnum.cu"

namespace cuFIXNUM {

/*
 * This is an archetypal implementation of a fixnum instruction
 * set. It defines the de facto interface for such implementations.
 *
 * All methods are defined for the device. It is someone else's
 * problem to get the data onto the device.
 */
template< int BYTES_, typename digit_ = u32_fixnum >
class warp_fixnum {
public:
    // NB: Language convention: Call something a 'digit' when it is constant
    // across the slot, and call it a 'fixnum' when it can vary between lanes in
    // the slot. Similarly, prefix a function call with 'digit::' when the
    // arguments are interpreted component-wise, and with 'fixnum::' when
    // they're interpreted "across the slot".
    typedef digit_ digit;
    typedef warp_fixnum fixnum;

    static constexpr int BYTES = BYTES_;
    static constexpr int BITS = 8 * BYTES;
    static constexpr int SLOT_WIDTH = BYTES / digit::BYTES;
    typedef slot_layout<digit, SLOT_WIDTH> layout;

    static_assert(BYTES > 0,
                  "Fixnum bytes must be positive.");
    static_assert(BYTES % digit::BYTES == 0,
                  "Fixnum digit size must divide fixnum bytes.");
    // TODO: Specialise std::is_integral for fixnum_u32?
    //static_assert(std::is_integral< digit >::value,
    //        "digit must be integral.");

private:
    digit x;

    // TODO: These should be private
public:
    __device__ __forceinline__
    operator digit () const { return x; }

    __device__ __forceinline__
    operator digit &() { return x; }

public:
    __device__ __forceinline__
    warp_fixnum() { }

    // TODO: Shouldn't this be equivalent to the digit_to_fixnum() function
    // below?
    __device__ __forceinline__
    warp_fixnum(digit z) : x(z) { }

    /***************************
     * Representation functions.
     */

    /*
     * Set r using bytes, interpreting bytes as a base-256 unsigned
     * integer. Return the number of bytes used. If nbytes >
     * BYTES, then the last nbytes - BYTES are ignored.
     *
     * NB: Normally we would expect from_bytes to be exclusively a
     * device function, but it's the same for the host, so we leave it
     * in.
     */
    __host__ __device__ static int from_bytes(uint8_t *r, const uint8_t *bytes, int nbytes) {
        int n = min(nbytes, BYTES);
        memcpy(r, bytes, n);
        memset(r + n, 0, BYTES - n);
        return n;
    }

    /*
     * Set bytes using r, converting r to a base-256 unsigned
     * integer. Return the number of bytes written. If nbytes <
     * BYTES, then the last BYTES - nbytes are ignored.
     *
     * NB: Normally we would expect from_bytes to be exclusively a
     * device function, but it's the same for the host, so we leave it
     * in.
     */
    __host__ __device__ static int to_bytes(uint8_t *bytes, int nbytes, const uint8_t *r) {
        int n = min(nbytes, BYTES);
        memcpy(bytes, r, n);
        return n;
    }

    /*
     * Return digit at index idx.
     */
    __device__ static digit get(fixnum var, int idx) {
        return layout::shfl(var, idx);
    }

    /*
     * Set var digit at index idx to be x.
     */
    __device__ static void set(fixnum &var, digit x, int idx) {
        var = (layout::laneIdx() == idx) ? (fixnum)x : var;
    }

    /*
     * Return digit in most significant place. Might be zero.
     */
    __device__ static digit top_digit(fixnum var) {
        return layout::shfl(var, layout::toplaneIdx);
    }

    /*
     * Return digit in the least significant place. Might be zero.
     *
     * TODO: Not clear how to interpret this function with more exotic fixnum
     * implementations such as RNS.
     */
    __device__ static digit bottom_digit(fixnum var) {
        return layout::shfl(var, 0);
    }

    /***********************
     * Arithmetic functions.
     */

    // TODO: Handle carry in
    // TODO: A more consistent syntax might be
    // fixnum add(fixnum a, fixnum b)
    // fixnum add_cc(fixnum a, fixnum b, int &cy_out)
    // fixnum addc(fixnum a, fixnum b, int cy_in)
    // fixnum addc_cc(fixnum a, fixnum b, int cy_in, int &cy_out)
    __device__ static void add_cy(fixnum &r, digit &cy_hi, fixnum a, fixnum b) {
        digit cy;
        digit::add_cy(r, cy, a, b);
        // r propagates carries iff r = FIXNUM_MAX
        digit r_cy = effective_carries(cy_hi, digit::is_max(r), cy);
        digit::add(r, r, r_cy);
    }

    __device__ static void add(fixnum &r, fixnum a, fixnum b) {
        digit cy;
        add_cy(r, cy, a, b);
    }

    // TODO: Handle borrow in
    __device__ static void sub_br(fixnum &r, digit &br_hi, fixnum a, fixnum b) {
        digit br;
        digit::sub_br(r, br, a, b);
        // r propagates borrows iff r = FIXNUM_MIN
        digit r_br = effective_carries(br_hi, digit::is_min(r), br);
        digit::sub(r, r, r_br);
    }

    __device__ static void sub(fixnum &r, fixnum a, fixnum b) {
        digit br;
        sub_br(r, br, a, b);
    }

    __device__ static fixnum zero() {
        return digit::zero();
    }

    __device__ static fixnum one() {
        return digit(layout::laneIdx() == 0);
    }

    __device__ static fixnum two() {
        return digit(layout::laneIdx() == 0 ? 2 : 0);
    }

    __device__ static int is_zero(fixnum a) {
        return nonzero_mask(a) == 0;
    }

    __device__ static digit incr_cy(fixnum &r) {
        digit cy;
        add_cy(r, cy, r, one());
        return cy;
    }

    __device__ static digit decr_br(fixnum &r) {
        digit br;
        sub_br(r, br, r, one());
        return br;
    }

    __device__ static void neg(fixnum &r, fixnum a) {
        sub(r, zero(), a);
    }

    /*
     * r = a * u, where a is interpreted as a single word, and u a
     * full fixnum. a should be constant across the slot for the
     * result to make sense.
     *
     * TODO: Can this be refactored with mad_cy?
     * TODO: Come up with a better name for this function. It's
     * scalar multiplication in the vspace of polynomials...
     */
    __device__ static digit mul_digit(fixnum &r, digit a, fixnum u) {
        fixnum hi, lo;
        digit cy, cy_hi;

        digit::mul_wide(hi, lo, a, u);
        cy_hi = top_digit(hi);
        hi = layout::shfl_up0(hi, 1);
        add_cy(lo, cy, lo, hi);

        return cy_hi + cy;
    }

    /*
     * r = lo_half(a * b)
     *
     * The "lo_half" is the product modulo 2^(8*BYTES),
     * i.e. the same size as the inputs.
     */
    __device__ static void mul_lo(fixnum &r, fixnum a, fixnum b) {
        // TODO: Implement specific mul_lo function.
        digit cy = digit::zero();

        r = zero();
        for (int i = layout::WIDTH - 1; i >= 0; --i) {
            digit aa = layout::shfl(a, i);

            digit::mad_hi_cy(r, cy, aa, b, r);
            // TODO: Could use rotate here, which is slightly
            // cheaper than shfl_up0...
            r = layout::shfl_up0(r, 1);
            cy = layout::shfl_up0(cy, 1);
            digit::mad_lo_cy(r, cy, aa, b, r);
        }
        cy = layout::shfl_up0(cy, 1);
        add(r, r, cy);
    }

    /*
     * (s, r) = a * b
     *
     * r is the "lo half" (see mul_lo above) and s is the
     * corresponding "hi half".
     */
    __device__ static void mul_wide(fixnum &ss, fixnum &rr, fixnum a, fixnum b) {
        int L = layout::laneIdx();

        fixnum r, s;
        r = fixnum::zero();
        s = fixnum::zero();
        digit cy = digit::zero();

        fixnum ai = get(a, 0);
        digit::mul_lo(s, ai, b);
        r = L == 0 ? s : r;  // r[0] = s[0];
        s = layout::shfl_down0(s, 1);
        digit::mad_hi_cy(s, cy, ai, b, s);

        for (int i = 1; i < layout::WIDTH; ++i) {
            fixnum ai = get(a, i);
            digit::mad_lo_cc(s, ai, b, s);

            fixnum s0 = get(s, 0);
            r = (L == i) ? s0 : r; // r[i] = s[0]
            s = layout::shfl_down0(s, 1);

            // TODO: Investigate whether deferring this carry resolution until
            // after the loop improves performance much.
            digit::addc_cc(s, s, cy);  // add carry from prev digit
            digit::addc(cy, 0, 0);     // cy = CC.CF
            digit::mad_hi_cy(s, cy, ai, b, s);
        }
        cy = layout::shfl_up0(cy, 1);
        add(s, s, cy);
        rr = r;
        ss = s;
    }

    __device__ static void mul_hi(fixnum &s, fixnum a, fixnum b) {
        // TODO: Implement specific mul_hi function.
        fixnum r;
        mul_wide(s, r, a, b);
    }

    /*
     * Adapt "rediagonalisation" trick described in Figure 4 of Ozturk,
     * Guilford, Gopal (2013) "Large Integer Squaring on Intel
     * Architecture Processors".
     *
     * TODO: This function is only definitively faster than mul_wide when WIDTH
     * is 32 (but in that case it's ~50% faster).
     */
    __device__ static void
    sqr_wide_(fixnum &ss, fixnum &rr, fixnum a)
    {
        constexpr int W = layout::WIDTH;
        int L = layout::laneIdx();

        fixnum r, s;
        r = fixnum::zero();
        s = fixnum::zero();
        fixnum diag_lo = fixnum::zero();
        digit cy = digit::zero();

        for (int i = 0; i < W / 2; ++i) {
            fixnum a1, a2, s0;
            int lpi = L + i;
            // TODO: Explain how on Earth these formulae pick out the correct
            // terms for the squaring.
            // NB: Could achieve the same with iterative shuffle's; the expressions
            // would be clearer, but the shuffles would (presumably) be more expensive.
            a1 = get(a, lpi < W ? i : lpi - W/2);
            a2 = get(a, lpi < W ? lpi : W/2 + i);

            assert(L != 0 || digit::cmp(a1,a2)==0); // a1 = a2 when L == 0

            fixnum hi, lo;
            digit::mul_wide(hi, lo, a1, a2);

            // TODO: These two (almost identical) blocks cause lots of pipeline
            // stalls; need to find a way to reduce their data dependencies.
            digit::add_cyio(s, cy, s, lo);
            lo = get(lo, 0);
            diag_lo = (L == 2*i) ? lo : diag_lo;
            s0 = get(s, 0);
            r = (L == 2*i) ? s0 : r; // r[2i] = s[0]
            s = layout::shfl_down0(s, 1);

            digit::add_cyio(s, cy, s, hi);
            hi = get(hi, 0);
            diag_lo = (L == 2*i + 1) ? hi : diag_lo;
            s0 = get(s, 0);
            r = (L == 2*i + 1) ? s0 : r; // r[2i+1] = s[0]
            s = layout::shfl_down0(s, 1);
        }

        // TODO: All these carries and borrows into s should be accumulated into
        // one call.
        add(s, s, cy);

        fixnum overflow;
        lshift_small(s, s, 1);  // s *= 2
        lshift_small(r, overflow, r, 1);  // r *= 2
        add_cy(s, cy, s, overflow); // really a logior, since s was just lshifted.
        assert(digit::is_zero(cy));

        // Doubling r above means we've doubled the diagonal terms, though they
        // shouldn't be. Compensate by subtracting a copy of them here.
        digit br;
        sub_br(r, br, r, diag_lo);
        br = (L == 0) ? br : digit::zero();
        sub(s, s, br);

        // TODO: This is wasteful, since the odd lane lo's are discarded as are
        // the even lane hi's.
        fixnum lo, hi, ai = get(a, W/2 + L/2);
        digit::mul_lo(lo, ai, ai);
        digit::mul_hi(hi, ai, ai);
        fixnum diag_hi = L & 1 ? hi : lo;

        add(s, s, diag_hi);

        rr = r;
        ss = s;
    }

    __device__ __forceinline__ static void
    sqr_wide(fixnum &ss, fixnum &rr, fixnum a) {
        // Width below which the general multiplication function is used instead
        // of this one. TODO: 16 is very high; need to work out why we're not
        // doing better on smaller widths.
        constexpr int SQUARING_WIDTH_THRESHOLD = 16;
        if (layout::WIDTH < SQUARING_WIDTH_THRESHOLD)
            mul_wide(ss, rr, a, a);
        else
            sqr_wide_(ss, rr, a);
    }

    __device__ static void sqr_lo(fixnum &r, fixnum a) {
        // TODO: Implement specific sqr_lo function.
        fixnum s;
        sqr_wide(s, r, a);
    }

    __device__ static void sqr_hi(fixnum &s, fixnum a) {
        // TODO: Implement specific sqr_hi function.
        fixnum r;
        sqr_wide(s, r, a);
    }

    /*
     * Return a mask of width bits whose ith bit is set if and only if
     * the ith digit of r is nonzero. In particular, result is zero
     * iff r is zero.
     */
    __device__ static uint32_t nonzero_mask(fixnum r) {
        return layout::ballot( ! digit::is_zero(r));
    }

    /*
     * Return -1, 0, or 1, depending on whether x is less than, equal
     * to, or greater than y.
     */
    __device__ static int cmp(fixnum x, fixnum y) {
        fixnum r;
        digit br;
        sub_br(r, br, x, y);
        // r != 0 iff x != y. If x != y, then br != 0 => x < y.
        return nonzero_mask(r) ? (br ? -1 : 1) : 0;
    }

    /*
     * Return the index of the most significant digit of x, or -1 if x is
     * zero.
     */
    __device__ static int most_sig_dig(fixnum x) {
        // FIXME: Should be able to get this value from limits or numeric_limits
        // or whatever.
        enum { UINT32_BITS = 8 * sizeof(uint32_t) };
        static_assert(UINT32_BITS == 32, "uint32_t isn't 32 bits");

        uint32_t a = nonzero_mask(x);
        return UINT32_BITS - (internal::clz(a) + 1);
    }

    /*
     * Return the index of the most significant bit of x, or -1 if x is
     * zero.
     *
     * TODO: Give this function a better name; maybe floor_log2()?
     */
    __device__ static int msb(fixnum x) {
        int b = most_sig_dig(x);
        if (b < 0) return b;
        digit y = layout::shfl(x, b);
        // TODO: These two lines are basically the same as most_sig_dig();
        // refactor.
        int c = digit::clz(y);
        return digit::BITS - (c + 1) + digit::BITS * b;
    }

    /*
     * Return the 2-valuation of x, i.e. the integer k >= 0 such that
     * 2^k divides x but 2^(k+1) does not divide x.  Depending on the
     * representation, can think of this as CTZ(x) ("Count Trailing
     * Zeros").  The 2-valuation of zero is *ahem* fixnum::BITS.
     *
     * TODO: Refactor common code between here, msb() and
     * most_sig_dig(). Perhaps write msb in terms of two_valuation?
     *
     * FIXME: Pretty sure this function is broken; e.g. if x is 0 but width <
     * warpSize, the answer is wrong.
     */
    __device__ static int two_valuation(fixnum x) {
        uint32_t a = nonzero_mask(x);
        int b = internal::ctz(a), c = 0;
        if (b < SLOT_WIDTH) {
            digit y = layout::shfl(x, b);
            c = digit::ctz(y);
        } else
            b = SLOT_WIDTH;
        return c + b * digit::BITS;
    }

    __device__
    static void
    lshift_small(fixnum &y, fixnum &overflow, fixnum x, int b) {
        assert(b >= 0);
        assert(b <= digit::BITS);
        int L = layout::laneIdx();

        fixnum cy;
        digit::lshift(y, cy, x, b);
        overflow = top_digit(cy);
        overflow = (L == 0) ? overflow : fixnum::zero();
        cy = layout::shfl_up0(cy, 1);
        digit::add(y, y, cy); // logior
    }

    __device__
    static void
    lshift_small(fixnum &y, fixnum x, int b) {
        assert(b >= 0);
        assert(b <= digit::BITS);

        fixnum cy;
        digit::lshift(y, cy, x, b);
        cy = layout::shfl_up0(cy, 1);
        digit::add(y, y, cy); // logior
    }

    /*
     * Set y to be x shifted by b bits to the left; effectively
     * multiply by 2^b. Return the top b bits of x in overflow.
     *
     * FIXME: Currently assumes that fixnum is unsigned.
     *
     * TODO: Think of better names for these functions. Something like
     * mul_2exp.
     *
     * TODO: Could improve performance significantly by using the funnel shift
     * instruction: https://docs.nvidia.com/cuda/parallel-thread-execution/#logic-and-shift-instructions-shf
     */
    __device__
    static void
    lshift(fixnum &y, fixnum &overflow, fixnum x, int b) {
        assert(b >= 0);
        assert(b <= BITS);
        int q = b / digit::BITS, r = b % digit::BITS;

        y = layout::rotate_up(x, q);
        // Hi bits of y[i] (=overflow) become the lo bits of y[(i+1) % width]
        digit::lshift(y, overflow, y, r);
        overflow = layout::rotate_up(overflow, 1);
        // TODO: This was "y |= overflow"; any advantage to using logior?
        digit::add(y, y, overflow);

        fixnum t;
        int L = layout::laneIdx();
        digit::set_if(overflow, y, L <= q);  // Kill high (q-1) words of y;
        digit::rem_2exp(t, overflow, r);     // Kill high BITS - r bits of overflow[q]
        set(overflow, t, q);
        digit::set_if(y, y, L >= q);         // Kill low q words of y;
        digit::rshift(t, y, r);              // Kill low r bits of y[q]
        digit::lshift(t, t, r);
        set(y, t, q);
    }

    __device__
    static void
    lshift(fixnum &y, fixnum x, int b) {
        assert(b >= 0);
        assert(b <= BITS);
        int q = b / digit::BITS, r = b % digit::BITS;

        y = layout::shfl_up0(x, q);
        lshift_small(y, y, r);
    }

    /*
     * Set y to be x shifted by b bits to the right; effectively
     * divide by 2^b. Return the bottom b bits of x.
     *
     * TODO: Think of better names for these functions. Something like
     * mul_2exp.
     */
    __device__
    static void
    rshift(fixnum &y, fixnum &underflow, fixnum x, int b) {
        lshift(underflow, y, x, BITS - b);
    }

    __device__
    static void
    rshift(fixnum &y, fixnum x, int b) {
        fixnum underflow;
        rshift(y, underflow, x, b);
    }

private:
    __device__
    static void
    digit_to_fixnum(digit &c) {
        int L = layout::laneIdx();
        // TODO: Try without branching?  c &= -(digit)(L == 0);
        c = (L == 0) ? c : digit::zero();
    }

    __device__
    static digit
    effective_carries(digit &cy_hi, int propagate, int cy) {
        int L = layout::laneIdx();
        uint32_t allcarries, p, g;

        g = layout::ballot(cy);              // carry generate
        p = layout::ballot(propagate);       // carry propagate
        allcarries = (p | g) + g;                 // propagate all carries
        // NB: There is no way to unify these two expressions to remove the
        // conditional. The conditional should be optimised away though, since
        // WIDTH is a compile-time constant.
        cy_hi = (layout::WIDTH == WARPSIZE) // detect hi overflow
            ? (allcarries < g)
            : ((allcarries >> layout::WIDTH) & 1);
        allcarries = (allcarries ^ p) | (g << 1); // get effective carries
        return (allcarries >> L) & 1;
    }
};

} // End namespace cuFIXNUM
