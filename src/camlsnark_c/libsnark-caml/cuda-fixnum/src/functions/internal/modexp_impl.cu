#pragma once
#include <hip/hip_runtime.h>


namespace cuFIXNUM {

namespace internal
{
    /*
     * Return floor(log2(x)). In particular, if x = 2^b, return b.
     */
    __device__
    constexpr unsigned
    floorlog2(unsigned x) {
        return x == 1 ? 0 : 1 + floorlog2(x >> 1);
    }

    /*
     * The following function gives a reasonable choice of WINDOW_SIZE in the k-ary
     * modular exponentiation method for a fixnum of B = 2^b bytes.
     *
     * The origin of the table is as follows. The expected number of multiplications
     * for the k-ary method with n-bit exponent and d-bit window is given by
     *
     *   T(n, d) = 2^d - 2 + n - d + (n/d - 1)*(1 - 2^-d)
     *
     * (see Koç, C. K., 1995, "Analysis of Sliding Window Techniques for
     * Exponentiation", Equation 1). The following GP script calculates the values
     * of n at which the window size should increase (maximum n = 65536):
     *
     *   ? T(n,d) = 2^d - 2 + n - d + (n/d - 1) * (1 - 2^-d);
     *   ? M = [ vecsort([[n, d, T(n, d)*1.] | d <- [1 .. 16]], 3)[1][2] | n <- [1 .. 65536] ];
     *   ? maxd = M[65536]
     *   10
     *   ? [[d, vecmin([n | n <- [1 .. 65536], M[n] == d])] | d <- [1 .. maxd]]
     *   [[1, 1], [2, 7], [3, 35], [4, 122], [5, 369], [6, 1044], [7, 2823], [8, 7371], [9, 18726], [10, 46490]]
     *
     * Table entry i is the window size for a fixnum of 8*(2^i) bits (e.g. 512 =
     * 8*2^6 bits falls between 369 and 1044, so the window size is that of the
     * smaller, 369, so 5 is in place i = 6).
     */
    // NB: For some reason we're not allowed to put this table in the definition
    // of bytes_to_window_size().
    constexpr int BYTES_TO_K_ARY_WINDOW_SIZE_TABLE[] = {
       -1,
       -1, //bytes bits
        2, // 2^2    32
        3, // 2^3    64
        4, // 2^4   128
        4, // 2^5   256
        5, // 2^6   512
        5, // 2^7  1024
        6, // 2^8  2048
        7, // 2^9  4096
        8, //2^10  8192
        8, //2^11 16384
        9, //2^12 32768
        10,//2^13 65536
    };

    __device__
    constexpr int
    bytes_to_k_ary_window_size(unsigned bytes) {
        return BYTES_TO_K_ARY_WINDOW_SIZE_TABLE[floorlog2(bytes)];
    }


    /*
     * This Table 2 from Koç, C. K., 1995, "Analysis of Sliding Window
     * Techniques for Exponentiation".
     *
     * The resolution of this table is higher than the one above because it's
     * used in the fixed exponent modexp code and can benefit from using the
     * precise bit length of the exponent, whereas the table above has to
     * accommodate multiple different exponents simultaneously.
     */
    __constant__
    int BYTES_TO_CLNW_WINDOW_SIZE_TABLE[] = {
       -1, // bits
        4, //  128
        5, //  256
        5, //  384
        5, //  512
        6, //  640
        6, //  768
        6, //  896
        6, // 1024
        6, // 1152
        6, // 1280
        6, // 1408
        6, // 1536
        6, // 1664
        7, // 1792
        7, // 1920
        7, // 2048
    };

    __device__
    constexpr int
    bits_to_clnw_window_size(unsigned bits) {
        // The chained ternary condition is forced upon us by the Draconian
        // constraints of C++11 constexpr functions.
        return
            bits < 64 ? 2 :
            bits < 128 ? 3 :
            bits > 2048 ? 7 :
            BYTES_TO_CLNW_WINDOW_SIZE_TABLE[(bits / 8) / 16];
    }

} // End namespace internal

} // End namespace cuFIXNUM
