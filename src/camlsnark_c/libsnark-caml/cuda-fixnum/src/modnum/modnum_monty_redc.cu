#include "hip/hip_runtime.h"
#pragma once

#include "modnum/internal/monty.cu"

namespace cuFIXNUM {

template< typename fixnum_ >
class modnum_monty_redc {
public:
    typedef fixnum_ fixnum;
    typedef fixnum modnum;

    __device__ modnum_monty_redc(fixnum mod)
    : monty(mod) {
        if ( ! monty.is_valid) return;

        modinv<fixnum> minv;
        minv(inv_mod, mod, fixnum::BITS);
        fixnum::neg(inv_mod, inv_mod);
#ifndef NDEBUG
        fixnum tmp;
        fixnum::mul_lo(tmp, inv_mod, mod);
        fixnum::add(tmp, tmp, fixnum::one());
        assert(fixnum::is_zero(tmp));
#endif
    }

    __device__ modnum zero() const { return monty.zero(); }
    __device__ modnum one() const { return monty.one(); }
    __device__ void add(modnum &z, modnum x, modnum y) const { monty.add(z, x, y); }
    __device__ void sub(modnum &z, modnum x, modnum y) const { monty.sub(z, x, y); }
    __device__ void neg(modnum &z, modnum x, modnum y) const { monty.neg(z, x); }

    __device__ void sqr(modnum &z, modnum x) const {
        // FIXME: Fix this hack!
        z = zero();
        if (!monty.is_valid) return;

        modnum a_hi, a_lo;
        fixnum::sqr_wide(a_hi, a_lo, x);
        redc(z, a_hi, a_lo);
    }

    __device__ void mul(modnum &z, modnum x, modnum y) const {
        // FIXME: Fix this hack!
        z = zero();
        if (!monty.is_valid) return;

        modnum a_hi, a_lo;
        fixnum::mul_wide(a_hi, a_lo, x, y);
        redc(z, a_hi, a_lo);
    }

    // TODO: Might be worth specialising multiplication for this case, since one of
    // the operands is known.
    __device__ void to_modnum(modnum &z, fixnum x) const {
        mul(z, x, monty.Rsqr_mod);
    }

    __device__ void from_modnum(fixnum &z, modnum x) const {
        //mul(z, x, fixnum::one());
        redc(z, fixnum::zero(), x);
    }

private:
    internal::monty<fixnum> monty;
    // inv_mod * mod = -1 % 2^fixnum::BITS.
    fixnum inv_mod;

    __device__ void redc(fixnum &r, fixnum a_hi, fixnum a_lo) const;
};


template< typename fixnum >
__device__ void
modnum_monty_redc<fixnum>::redc(fixnum &r, fixnum a_hi, fixnum a_lo) const {
    typedef typename fixnum::digit digit;
    fixnum b, s_hi, s_lo;
    digit cy, c;

    // FIXME: Fix this hack!
    r = zero();
    if (!monty.is_valid) return;

    fixnum::mul_lo(b, a_lo, inv_mod);

    // This section is essentially s = floor(mad_wide(b, mod, a) / R)

    // TODO: Can we employ the trick to avoid a multiplication because we
    // know b = am' (mod R)?
    fixnum::mul_wide(s_hi, s_lo, b, monty.mod);
    // TODO: Only want the carry; find a cheaper way to determine that
    // without doing the full addition.
    fixnum::add_cy(s_lo, cy, s_lo, a_lo);

    // TODO: The fact that we need to turn cy into a fixnum before using it in
    // arithmetic should be handled more cleanly. Also, this code is already in
    // the private function digit_to_fixnum() in ''warp_fixnum.cu'.
    int L = fixnum::layout::laneIdx();
    cy = (L == 0) ? cy : digit::zero();

    // TODO: The assert below fails; work out why.
#if 0
    // NB: b = am' (mod R) => a + bm = a + amm' = 2a (mod R). So surely
    // all I need to propagate is the top bit of a_lo?
    fixnum top_bit, dummy;
    fixnum::lshift(dummy, top_bit, a_lo, 1);
    assert(digit::cmp(cy, top_bit) == 0);
#endif
    fixnum::add_cy(r, cy, s_hi, cy);
    fixnum::add_cy(r, c, r, a_hi);
    digit::add(cy, cy, c);
    assert(cy == !!cy); // cy = 0 or 1.

    monty.normalise(r, cy);
}

} // End namespace cuFIXNUM
