#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>
#include <vector>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"

const unsigned int bytes_per_elem = 128;
const unsigned int io_bytes_per_elem = 96;


using namespace std;
using namespace cuFIXNUM;

template< typename fixnum >
struct mul_and_convert {
  // redc may be worth trying over cios
  typedef modnum_monty_cios<fixnum> modnum;
  __device__ void operator()(fixnum &r, fixnum a, fixnum b, fixnum my_mod) {
      modnum mod = modnum(my_mod);

      fixnum sm;
      mod.mul(sm, a, b);

      fixnum s;
      mod.from_modnum(s, sm);

      r = s;
  }
};

template< int fn_bytes, typename fixnum_array >
void print_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);

    for (int i = 0; i < lrl; i++) {
      printf("%i ", local_results[i]);
    }
    printf("\n");
}

template< int fn_bytes, typename fixnum_array >
vector<uint8_t*> get_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);
    vector<uint8_t*> res_v;
    for (int n = 0; n < nelts; n++) {
      uint8_t* a = (uint8_t*)malloc(fn_bytes*sizeof(uint8_t));
      for (int i = 0; i < fn_bytes; i++) {
        a[i] = local_results[n*fn_bytes + i];
      }
      res_v.emplace_back(a);
    }
    return res_v;
}


template< int fn_bytes, typename word_fixnum, template <typename> class Func >
std::vector<uint8_t*> compute_product(std::vector<uint8_t*> a, std::vector<uint8_t*> b, uint8_t* input_m_base) {
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    int nelts = a.size();

    uint8_t *input_a = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_a[i] = a[i/fn_bytes][i%fn_bytes];
    }

    uint8_t *input_b = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_b[i] = b[i/fn_bytes][i%fn_bytes];
    }

    uint8_t *input_m = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_m[i] = input_m_base[i%fn_bytes];
    }

    // TODO reuse modulus as a constant instead of passing in nelts times
    fixnum_array *res, *in_a, *in_b, *inM;
    in_a = fixnum_array::create(input_a, fn_bytes * nelts, fn_bytes);
    in_b = fixnum_array::create(input_b, fn_bytes * nelts, fn_bytes);
    inM = fixnum_array::create(input_m, fn_bytes * nelts, fn_bytes);
    res = fixnum_array::create(nelts);

    fixnum_array::template map<Func>(res, in_a, in_b, inM);

    vector<uint8_t*> v_res = get_fixnum_array<fn_bytes, fixnum_array>(res, nelts);

    //TODO to do stage 1 field arithmetic, instead of a map, do a reduce

    delete in_a;
    delete in_b;
    delete inM;
    delete res;
    delete[] input_a;
    delete[] input_b;
    delete[] input_m;
    return v_res;
}

uint8_t* read_mnt_fq(FILE* inputs) {
  uint8_t* buf = (uint8_t*)calloc(bytes_per_elem, sizeof(uint8_t));
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  fread((void*)( buf + (bytes_per_elem - io_bytes_per_elem)), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  return buf;
}

void write_mnt_fq(uint8_t* fq, FILE* outputs) {
  fwrite((void *) fq, io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
}

int do_fixnum_example(const char *inputs_file, const char *outputs_file) {
  setbuf(stdout, NULL);

  // mnt4_q
  uint8_t mnt4_modulus[bytes_per_elem] = {1,128,94,36,222,99,144,94,159,17,221,44,82,84,157,227,240,37,196,154,113,16,136,99,164,84,114,118,233,204,90,104,56,126,83,203,165,13,15,184,157,5,24,242,118,231,23,177,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  // mnt6_q
  uint8_t mnt6_modulus[bytes_per_elem] = {1,0,0,64,226,118,7,217,79,58,161,15,23,153,160,78,151,87,0,63,188,129,195,214,164,58,153,52,118,249,223,185,54,38,33,41,148,202,235,62,155,169,89,200,40,92,108,178,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  auto inputs = fopen(inputs_file, "r");
  auto outputs = fopen(outputs_file, "w");

  size_t n;

   while (true) {
    size_t elts_read = fread((void *) &n, sizeof(size_t), 1, inputs);
    if (elts_read == 0) { break; }

    std::vector<uint8_t*> x0;
    for (size_t i = 0; i < n/2; ++i) {
      x0.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> x1;
    for (size_t i = 0; i < n/2; ++i) {
      x1.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> res_x = compute_product<bytes_per_elem, u64_fixnum, mul_and_convert>(x0, x1, mnt4_modulus);

    for (size_t i = 0; i < n/2; ++i) {
      write_mnt_fq(res_x[i], outputs);
    }

    std::vector<uint8_t*> y0;
    for (size_t i = 0; i < n/2; ++i) {
      y0.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> y1;
    for (size_t i = 0; i < n/2; ++i) {
      y1.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> res_y = compute_product<bytes_per_elem, u64_fixnum, mul_and_convert>(y0, y1, mnt6_modulus);

    for (size_t i = 0; i < n/2; ++i) {
      write_mnt_fq(res_y[i], outputs);
    }

    for (size_t i = 0; i < n/2; ++i) {
      free(x0[i]);
      free(x1[i]);
      free(y0[i]);
      free(y1[i]);
      free(res_x[i]);
      free(res_y[i]);
    }

  }

  return 0;
}

