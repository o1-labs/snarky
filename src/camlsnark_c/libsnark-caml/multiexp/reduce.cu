#include "hip/hip_runtime.h"
#include <cstdint>
#include <vector>
#include <chrono>
#include <memory>
#include <hip/hip_cooperative_groups.h>

#include "curves.cu"

// C is the size of the precomputation
// R is the number of points we're handling per thread
template< typename EC, int C = 4, int RR = 8 >
__global__ void
ec_multiexp_straus(var *out, const var *multiples_, const var *scalars_, size_t N)
{
    int T = threadIdx.x, B = blockIdx.x, D = blockDim.x;
    int elts_per_block = D / BIG_WIDTH;
    int tileIdx = T / BIG_WIDTH;

    int idx = elts_per_block * B + tileIdx;

    size_t n = (N + RR - 1) / RR;
    if (idx < n) {
        // TODO: Treat remainder separately so R can remain a compile time constant
        size_t R = (idx < n - 1) ? RR : (N % RR);

        typedef typename EC::group_type Fr;
        static constexpr int JAC_POINT_LIMBS = 3 * EC::field_type::DEGREE * ELT_LIMBS;
        static constexpr int AFF_POINT_LIMBS = 2 * EC::field_type::DEGREE * ELT_LIMBS;
        int out_off = idx * JAC_POINT_LIMBS;
        int m_off = idx * RR * AFF_POINT_LIMBS;
        int s_off = idx * RR * ELT_LIMBS;

        Fr scalars[RR];
        for (int j = 0; j < R; ++j) {
            Fr::load(scalars[j], scalars_ + s_off + j*ELT_LIMBS);
            Fr::from_monty(scalars[j], scalars[j]);
        }

        const var *multiples = multiples_ + m_off;
        // TODO: Consider loading multiples and/or scalars into shared memory

        // i is smallest multiple of C such that i > 753
        int i = C * ((753 + C - 1) / C); // C * ceiling(753/C)
        assert((i - C * 753) < C);
        static constexpr var C_MASK = (1U << C) - 1U;

        EC x;
        EC::set_zero(x);
        while (i >= C) {
            EC::mul_2exp<C>(x, x);
            i -= C;

            int q = i / digit::BITS, r = i % digit::BITS;
            for (int j = 0; j < R; ++j) {
                //(scalars[j][q] >> r) & C_MASK
                auto g = fixnum::layout();
                var s = g.shfl(scalars[j].a, q);
                var win = (s >> r) & C_MASK;
                // Handle case where C doesn't divide digit::BITS
                int bottom_bits = digit::BITS - r;
                // detect when window overlaps digit boundary
                if (bottom_bits < C) {
                    s = g.shfl(scalars[j].a, q + 1);
                    win |= (s << bottom_bits) & C_MASK;
                }
                if (win > 0) {
                    EC m;
                    //EC::add(x, x, multiples[win - 1][j]);
                    EC::load_affine(m, multiples + ((win-1)*N + j)*AFF_POINT_LIMBS);
                    EC::mixed_add(x, x, m);
                }
            }
        }
        EC::store_jac(out + out_off, x);
    }
}

template< typename EC >
__global__ void
ec_multiexp(var *X, const var *W, size_t n)
{
    int T = threadIdx.x, B = blockIdx.x, D = blockDim.x;
    int elts_per_block = D / BIG_WIDTH;
    int tileIdx = T / BIG_WIDTH;

    int idx = elts_per_block * B + tileIdx;

    if (idx < n) {
        typedef typename EC::group_type Fr;
        EC x;
        Fr w;
        int x_off = idx * EC::NELTS * ELT_LIMBS;
        int w_off = idx * ELT_LIMBS;

        EC::load_affine(x, X + x_off);
        Fr::load(w, W + w_off);

        // We're given W in Monty form for some reason, so undo that.
        Fr::from_monty(w, w);
        EC::mul(x, w.a, x);

        EC::store_jac(X + x_off, x);
    }
}

template< typename EC >
__global__ void
ec_sum_all(var *X, const var *Y, size_t n)
{
    int T = threadIdx.x, B = blockIdx.x, D = blockDim.x;
    int elts_per_block = D / BIG_WIDTH;
    int tileIdx = T / BIG_WIDTH;

    int idx = elts_per_block * B + tileIdx;

    if (idx < n) {
        EC z, x, y;
        int off = idx * EC::NELTS * ELT_LIMBS;

        EC::load_jac(x, X + off);
        EC::load_jac(y, Y + off);

        EC::add(z, x, y);

        EC::store_jac(X + off, z);
    }
}

static constexpr size_t threads_per_block = 256;

template< typename EC, int C, int R >
void
ec_reduce_straus(hipStream_t &strm, var *out, const var *multiples, const var *scalars, size_t N)
{
    hipStreamCreate(&strm);

    static constexpr size_t pt_limbs = EC::NELTS * ELT_LIMBS;
    size_t n = (N + R - 1) / R;

    size_t nblocks = (n * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

    ec_multiexp_straus<EC, C, R><<< nblocks, threads_per_block, 0, strm>>>(out, multiples, scalars, N);

    size_t r = n & 1, m = n / 2;
    for ( ; m != 0; r = m & 1, m >>= 1) {
        nblocks = (m * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

        ec_sum_all<EC><<<nblocks, threads_per_block, 0, strm>>>(out, out + m*pt_limbs, m);
        if (r)
            ec_sum_all<EC><<<1, threads_per_block, 0, strm>>>(out, out + 2*m*pt_limbs, 1);
    }
}

template< typename EC >
void
ec_reduce(hipStream_t &strm, var *X, const var *w, size_t n)
{
    hipStreamCreate(&strm);

    size_t nblocks = (n * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

    // FIXME: Only works on Pascal and later.
    //auto grid = cg::this_grid();
    ec_multiexp<EC><<< nblocks, threads_per_block, 0, strm>>>(X, w, n);

    static constexpr size_t pt_limbs = EC::NELTS * ELT_LIMBS;

    size_t r = n & 1, m = n / 2;
    for ( ; m != 0; r = m & 1, m >>= 1) {
        nblocks = (m * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

        ec_sum_all<EC><<<nblocks, threads_per_block, 0, strm>>>(X, X + m*pt_limbs, m);
        if (r)
            ec_sum_all<EC><<<1, threads_per_block, 0, strm>>>(X, X + 2*m*pt_limbs, 1);
        // TODO: Not sure this is really necessary.
        //grid.sync();
    }
}

static inline double as_mebibytes(size_t n) {
    return n / (long double)(1UL << 20);
}

void print_meminfo(size_t allocated) {
    size_t free_mem, dev_mem;
    hipMemGetInfo(&free_mem, &dev_mem);
    fprintf(stderr, "Allocated %zu bytes; device has %.1f MiB free (%.1f%%).\n",
            allocated,
            as_mebibytes(free_mem),
            100.0 * free_mem / dev_mem);
}

struct CudaFree {
    void operator()(var *mem) { hipFree(mem); }
};
typedef std::unique_ptr<var, CudaFree> var_ptr;

var_ptr
allocate_memory(size_t nbytes, int dbg = 0) {
    var *mem = nullptr;
    hipMallocManaged(&mem, nbytes);
    if (mem == nullptr) {
        fprintf(stderr, "Failed to allocate enough device memory\n");
        abort();
    }
    if (dbg)
        print_meminfo(nbytes);
    return var_ptr(mem);
}

var_ptr
load_scalars(size_t n, FILE *inputs)
{
    static constexpr size_t scalar_bytes = ELT_BYTES;
    size_t total_bytes = n * scalar_bytes;

    auto mem = allocate_memory(total_bytes);
    if (fread((void *)mem.get(), total_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read scalars\n");
        abort();
    }
    return mem;
}

template< typename EC >
var_ptr
load_points(size_t n, FILE *inputs)
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;
    static constexpr size_t jac_pt_bytes = 3 * coord_bytes;

    size_t total_aff_bytes = n * aff_pt_bytes;
    size_t total_jac_bytes = n * jac_pt_bytes;

    auto mem = allocate_memory(total_jac_bytes);
    if (fread((void *)mem.get(), total_aff_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read all curve poinst\n");
        abort();
    }

    // insert space for z-coordinates
    char *cmem = reinterpret_cast<char *>(mem.get()); //lazy
    for (size_t i = n - 1; i > 0; --i) {
        char tmp_pt[aff_pt_bytes];
        memcpy(tmp_pt, cmem + i * aff_pt_bytes, aff_pt_bytes);
        memcpy(cmem + i * jac_pt_bytes, tmp_pt, aff_pt_bytes);
    }
    return mem;
}

template< typename EC >
var_ptr
load_points_affine(size_t n, FILE *inputs)
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;

    size_t total_aff_bytes = n * aff_pt_bytes;

    auto mem = allocate_memory(total_aff_bytes);
    if (fread((void *)mem.get(), total_aff_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read all curve poinst\n");
        abort();
    }
    return mem;
}
